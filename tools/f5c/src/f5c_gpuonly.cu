/* @file f5c_gpuonly.cu
**
** implementation of the f5c GPU-only framework (opposed to the CPU-GPU hybrid approach in f5c.cu)
** not compiled by default
** @author: Hasindu Gamaarachchi (hasindu@unsw.edu.au)
** @@
******************************************************************************/

#include <errno.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include "error.h"
#include "f5c.h"
#include "f5cmisc.cuh"
#include "f5cmisc.h"


#ifndef CPU_GPU_PROC

/* if defined, static cuda/cpu arrays (model and arrays dependent on K)
   are preallocated at the beginning of the program, rather than repeatedly doing so inside a loop */
#define CUDA_PRE_MALLOC 1

void align_cuda(core_t* core, db_t* db) {
    int32_t i;
    int32_t n_bam_rec = db->n_bam_rec;
    double realtime1;

    /**cuda pointers*/
    char* read;        //flattened reads sequences
    ptr_t* read_ptr; //index pointer for flattedned "reads"
    int32_t* read_len;
    int64_t sum_read_len;
    int32_t* n_events;
    event_t* event_table;
    ptr_t* event_ptr;
    int64_t sum_n_events;
    scalings_t* scalings;
    AlignedPair* event_align_pairs;
    int32_t* n_event_align_pairs;
    float *bands;
    uint8_t *trace;
    EventKmerPair* band_lower_left;

realtime1 = realtime();

    int32_t cuda_device_num = core->opt.cuda_dev_id;
    hipSetDevice(cuda_device_num);
    CUDA_CHK();

#ifdef CUDA_PRE_MALLOC
    ptr_t* read_ptr_host = core->cuda->read_ptr_host;
#else
    //get the total size and create the pointers
    ptr_t* read_ptr_host = (ptr_t*)malloc(sizeof(ptr_t) * n_bam_rec);
    MALLOC_CHK(read_ptr_host);
#endif
    sum_read_len = 0;

    //read sequences : needflattening
    for (i = 0; i < n_bam_rec; i++) {
        read_ptr_host[i] = sum_read_len;
        sum_read_len += (db->read_len[i] + 1); //with null term
    }
    //form the temporary flattened array on host
    char* read_host = (char*)malloc(sizeof(char) * sum_read_len);
    MALLOC_CHK(read_host);
    for (i = 0; i < n_bam_rec; i++) {
        ptr_t idx = read_ptr_host[i];
        strcpy(&read_host[idx], db->read[i]);
    }

    //now the events : need flattening
    //num events : need flattening
    //get the total size and create the pointers
#ifdef CUDA_PRE_MALLOC
    int32_t* n_events_host = core->cuda->n_events_host;
    ptr_t* event_ptr_host = core->cuda->event_ptr_host;
#else
    int32_t* n_events_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(n_events_host);
    ptr_t* event_ptr_host = (ptr_t*)malloc(sizeof(ptr_t) * n_bam_rec);
    MALLOC_CHK(event_ptr_host);
#endif

    sum_n_events = 0;
    for (i = 0; i < n_bam_rec; i++) {
        n_events_host[i] = db->et[i].n;
        event_ptr_host[i] = sum_n_events;
        sum_n_events += db->et[i].n;
    }

    //event table flatten
    //form the temporary flattened array on host
    event_t* event_table_host =
        (event_t*)malloc(sizeof(event_t) * sum_n_events);
    MALLOC_CHK(event_table_host);
    for (i = 0; i < n_bam_rec; i++) {
        ptr_t idx = event_ptr_host[i];
        memcpy(&event_table_host[idx], db->et[i].event,
               sizeof(event_t) * db->et[i].n);
    }

    AlignedPair* event_align_pairs_host =
        (AlignedPair*)malloc(2 * sum_n_events * sizeof(AlignedPair));
    MALLOC_CHK(event_align_pairs_host);

core->align_cuda_preprocess += (realtime() - realtime1);

    /** Start GPU mallocs**/
realtime1 = realtime();

#ifdef CUDA_PRE_MALLOC
    read_ptr =core->cuda->read_ptr;
    read_len=core->cuda->read_len;
    n_events=core->cuda->n_events;
    event_ptr=core->cuda->event_ptr;
    scalings=core->cuda->scalings;
    model_t* model = core->cuda->model;
#else

    if(core->opt.verbosity>1) print_size("read_ptr array",n_bam_rec * sizeof(ptr_t));
    hipMalloc((void**)&read_ptr, n_bam_rec * sizeof(ptr_t));
    CUDA_CHK();

    if(core->opt.verbosity>1) print_size("read_lens",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&read_len, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //n_events
    if(core->opt.verbosity>1) print_size("n_events",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&n_events, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //event ptr
    if(core->opt.verbosity>1) print_size("event ptr",n_bam_rec * sizeof(ptr_t));
    hipMalloc((void**)&event_ptr, n_bam_rec * sizeof(ptr_t));
    CUDA_CHK();
    //scalings : already linear
    if(core->opt.verbosity>1) print_size("Scalings",n_bam_rec * sizeof(scalings_t));
    hipMalloc((void**)&scalings, n_bam_rec * sizeof(scalings_t));
    CUDA_CHK();
    //model : already linear
    model_t* model;
    hipMalloc((void**)&model,
            MAX_NUM_KMER * sizeof(model_t));
    CUDA_CHK();
#endif


    if(core->opt.verbosity>1) print_size("read array",sum_read_len * sizeof(char));
    hipMalloc((void**)&read, sum_read_len * sizeof(char)); //with null char
    CUDA_CHK();
    if(core->opt.verbosity>1) print_size("event table",sum_n_events * sizeof(event_t));
    hipMalloc((void**)&event_table, sum_n_events * sizeof(event_t));
    CUDA_CHK();
    model_t* model_kmer_cache;
    hipMalloc((void**)&model_kmer_cache, sum_read_len * sizeof(model_t));
    CUDA_CHK();

    /**allocate output arrays for cuda**/
    if(core->opt.verbosity>1) print_size("event align pairs",2 * sum_n_events *sizeof(AlignedPair));
    hipMalloc((void**)&event_align_pairs,
            2 * sum_n_events *
                sizeof(AlignedPair)); //todo : need better huristic
    CUDA_CHK();
#ifdef CUDA_PRE_MALLOC
    n_event_align_pairs=core->cuda->n_event_align_pairs;
#else
    if(core->opt.verbosity>1) print_size("n_event_align_pairs",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&n_event_align_pairs, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
#endif
    //scratch arrays
    size_t sum_n_bands = sum_n_events + sum_read_len; //todo : can be optimised
    if(core->opt.verbosity>1) print_size("bands",sizeof(float) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&bands,sizeof(float) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    if(core->opt.verbosity>1) print_size("trace",sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&trace, sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    hipMemset(trace,0,sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH); //initialise the trace array to 0
    if(core->opt.verbosity>1) print_size("band_lower_left",sizeof(EventKmerPair)* sum_n_bands);
    hipMalloc((void**)&band_lower_left, sizeof(EventKmerPair)* sum_n_bands);
    CUDA_CHK();
core->align_cuda_malloc += (realtime() - realtime1);

    /* cuda mem copys*/
realtime1 =realtime();
    hipMemcpy(read_ptr, read_ptr_host, n_bam_rec * sizeof(ptr_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(read, read_host, sum_read_len * sizeof(char),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    //read length : already linear hence direct copy
    hipMemcpy(read_len, db->read_len, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(n_events, n_events_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(event_ptr, event_ptr_host, n_bam_rec * sizeof(ptr_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(event_table, event_table_host, sizeof(event_t) * sum_n_events,
               hipMemcpyHostToDevice);
    CUDA_CHK();

#ifndef CUDA_PRE_MALLOC
//model : already linear //move to cuda_init
    hipMemcpy(model, core->model, MAX_NUM_KMER * sizeof(model_t),
            hipMemcpyHostToDevice);
    CUDA_CHK();
#endif
    //can be interleaved
    hipMemcpy(scalings, db->scalings, sizeof(scalings_t) * n_bam_rec,
               hipMemcpyHostToDevice);
    CUDA_CHK();
core->align_cuda_memcpy += (realtime() - realtime1);

    uint32_t kmer_size = core->kmer_size;

realtime1 = realtime();

    /*pre kernel*/
    assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
    dim3 gridpre(1,(db->n_bam_rec + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
    dim3 blockpre(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);
	if(core->opt.verbosity>1) fprintf(stderr,"grid %d,%d, block %d,%d\n",gridpre.x,gridpre.y, blockpre.x,blockpre.y);

    align_kernel_pre_2d<<<gridpre, blockpre>>>( read,
        read_len, read_ptr, n_events,
        event_ptr, model, kmer_size, n_bam_rec, model_kmer_cache,bands,trace,band_lower_left);

    hipDeviceSynchronize();CUDA_CHK();
    if(core->opt.verbosity>1) fprintf(stderr, "[%s::%.3f*%.2f] align-pre kernel done\n", __func__,
            realtime() - realtime1, cputime() / (realtime() - realtime1));
core->align_kernel_time += (realtime() - realtime1);
core->align_pre_kernel_time += (realtime() - realtime1);

realtime1 = realtime();

    /* core kernel*/
    assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
    dim3 grid1(1,(db->n_bam_rec + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
    dim3 block1(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);
    align_kernel_core_2d_shm<<<grid1, block1>>>(read_len, read_ptr, event_table, n_events,
            event_ptr, scalings, n_bam_rec, model_kmer_cache, kmer_size, bands,trace,band_lower_left );

    hipDeviceSynchronize();CUDA_CHK();
    if(core->opt.verbosity>1) fprintf(stderr, "[%s::%.3f*%.2f] align-core kernel done\n", __func__,
    realtime() - realtime1, cputime() / (realtime() - realtime1));
    core->align_kernel_time += (realtime() - realtime1);
core->align_core_kernel_time += (realtime() - realtime1);

realtime1 = realtime();

    /*post kernel*/
    int32_t BLOCK_LEN = core->opt.cuda_block_size;
    dim3 gridpost((db->n_bam_rec + BLOCK_LEN - 1) / BLOCK_LEN);
    dim3 blockpost(BLOCK_LEN);
    #ifndef WARP_HACK
        align_kernel_post<<<gridpost, blockpost>>>(event_align_pairs, n_event_align_pairs,
            read_len, read_ptr, event_table, n_events,
            event_ptr,scalings, n_bam_rec, model_kmer_cache, kmer_size, bands,trace,band_lower_left );

    #else
        assert(BLOCK_LEN>=32);
        dim3 grid1post((db->n_bam_rec + (BLOCK_LEN/32) - 1) / (BLOCK_LEN/32));
        if(core->opt.verbosity>1) fprintf(stderr,"grid new %d\n",grid1post.x);
        align_kernel_post<<<grid1post, blockpost>>>(event_align_pairs, n_event_align_pairs,
            read_len, read_ptr, event_table, n_events,
            event_ptr, scalings, n_bam_rec, model_kmer_cache, kmer_size, bands,trace,band_lower_left );
    #endif
    hipDeviceSynchronize();CUDA_CHK();
    if(core->opt.verbosity>1) fprintf(stderr, "[%s::%.3f*%.2f] align-post kernel done\n", __func__,
            realtime() - realtime1, cputime() / (realtime() - realtime1));
    core->align_kernel_time += (realtime() - realtime1);
core->align_post_kernel_time += (realtime() - realtime1);


    //fprintf(stderr,"readlen %d,n_events %d\n",db->read_len[i],n_event_align_pairs);

#ifdef CUDA_DEBUG

    hipDeviceSynchronize();
    CUDA_CHK();

#endif

    /** copyback ans**/
realtime1 =  realtime();
    hipMemcpy(db->n_event_align_pairs, n_event_align_pairs,
               n_bam_rec * sizeof(int32_t), hipMemcpyDeviceToHost);
    CUDA_CHK();

    hipMemcpy(event_align_pairs_host, event_align_pairs,
               2 * sum_n_events * sizeof(AlignedPair), hipMemcpyDeviceToHost);
    CUDA_CHK();
core->align_cuda_memcpy += (realtime() - realtime1);

realtime1 =  realtime();
#ifndef CUDA_PRE_MALLOC
    hipFree(read_ptr);
    hipFree(read_len);
    hipFree(n_events);
    hipFree(event_ptr);
    hipFree(model); //constant memory
    hipFree(scalings);
    hipFree(n_event_align_pairs);
#endif
    hipFree(read); //with null char
    hipFree(event_table);
    hipFree(event_align_pairs);
    hipFree(bands);
    hipFree(trace);
    hipFree(band_lower_left);
    hipFree(model_kmer_cache);

core->align_cuda_malloc += (realtime() - realtime1);

    /** post work**/
realtime1 =  realtime();
    //copy back
    for (i = 0; i < n_bam_rec; i++) {
        ptr_t idx = event_ptr_host[i];
        memcpy(db->event_align_pairs[i], &event_align_pairs_host[idx * 2],
               sizeof(AlignedPair) * db->n_event_align_pairs[i]);
    }

    //free the temp arrays on host
#ifndef CUDA_PRE_MALLOC
    free(read_ptr_host);
    free(n_events_host);
    free(event_ptr_host);
#endif
    free(read_host);
    free(event_table_host);
    free(event_align_pairs_host);


core->align_cuda_postprocess += (realtime() - realtime1);

}


#endif
