#include "hip/hip_runtime.h"
/* @file align.cu
**
**  GPU implementation of the Adaptive banded Event Alignment algorithm
** @author: Hasindu Gamaarachchi (hasindu@unsw.edu.au)
** @@
******************************************************************************/

#include "f5c.h"
#include <assert.h>
#include "f5cmisc.cuh"

//#define DEBUG_ESTIMATED_SCALING 1
//#define DEBUG_RECALIB_SCALING 1
//#define DEBUG_ADAPTIVE 1

//todo : performing __sync_threads inside the loops is not ideal. Works for today's CUDA architectures.
// If kernels hang in a future CUDA architecture, this may be the culprit

//todo : can make more efficient using bit encoding
//todo : is inlining correct?
__forceinline__ __device__  uint32_t get_rank(char base) {
    if (base == 'A') { //todo: do we neeed simple alpha?
        return 0;
    } else if (base == 'C') {
        return 1;
    } else if (base == 'G') {
        return 2;
    } else if (base == 'T') {
        return 3;
    } else {
        //WARNING("A None ACGT base found : %c", base); //todo : fix this in gpu code
        return 0;
    }
}

// return the lexicographic rank of the kmer amongst all strings of
// length k for this alphabet
__forceinline__ __device__  uint32_t get_kmer_rank(const char* str, uint32_t k) {
    //uint32_t p = 1;
    uint32_t r = 0;

    // from last base to first
    for (uint32_t i = 0; i < k; ++i) {
        //r += rank(str[k - i - 1]) * p;
        //p *= size();
        r += get_rank(str[k - i - 1]) << (i << 1);
    }
    return r;
}

//copy a kmer from a reference
__forceinline__ __device__ void kmer_cpy(char* dest, char* src, uint32_t k) {
    uint32_t i = 0;
    for (i = 0; i < k; i++) {
        dest[i] = src[i];
    }
    dest[i] = '\0';
}

#define log_inv_sqrt_2pi  -0.918938f // Natural logarithm

__forceinline__ __device__ float
log_normal_pdf(float x, float gp_mean, float gp_stdv, float gp_log_stdv) {
    /*INCOMPLETE*/
    //float log_inv_sqrt_2pi = -0.918938f; // Natural logarithm
    float a = (x - gp_mean) / gp_stdv;
    return log_inv_sqrt_2pi - gp_log_stdv + (-0.5f * a * a);
    // return 1;
}

__forceinline__ __device__ float
log_probability_match_r9(scalings_t scaling, model_t* models, event_t* event,
                         int event_idx, uint32_t kmer_rank) {
    // event level mean, scaled with the drift value
    //strand = 0;
 #ifdef DEBUG_ADAPTIVE
    assert(kmer_rank < 4096);
 #endif
    //float level = read.get_drift_scaled_level(event_idx, strand);

    //float time =
    //    (events.event[event_idx].start - events.event[0].start) / sample_rate;
    float unscaledLevel = event[event_idx].mean;
    float scaledLevel = unscaledLevel;
    //float scaledLevel = unscaledLevel - time * scaling.shift;

    //fprintf(stderr, "level %f\n",scaledLevel);
    //GaussianParameters gp = read.get_scaled_gaussian_from_pore_model_state(pore_model, strand, kmer_rank);
    float gp_mean =
        scaling.scale * models[kmer_rank].level_mean + scaling.shift;
    float gp_stdv = models[kmer_rank].level_stdv * 1; //scaling.var = 1;
    // float gp_stdv = 0;
    // float gp_log_stdv = models[kmer_rank].level_log_stdv + scaling.log_var;
    // if(models[kmer_rank].level_stdv <0.01 ){
    // 	fprintf(stderr,"very small std dev %f\n",models[kmer_rank].level_stdv);
    // }
    #ifdef CACHED_LOG
        float gp_log_stdv = models[kmer_rank].level_log_stdv;
    #else
        float gp_log_stdv =
        log(models[kmer_rank].level_stdv); // scaling.log_var = log(1)=0;
    #endif

    float lp = log_normal_pdf(scaledLevel, gp_mean, gp_stdv, gp_log_stdv);
    return lp;
}

#define event_kmer_to_band(ei, ki) (ei + 1) + (ki + 1)
#define band_event_to_offset(bi, ei) band_lower_left[bi].event_idx - (ei)
#define band_kmer_to_offset(bi, ki) (ki) - band_lower_left[bi].kmer_idx
#define is_offset_valid(offset) (offset) >= 0 && (offset) < bandwidth
#define event_at_offset(bi, offset) band_lower_left[(bi)].event_idx - (offset)
#define kmer_at_offset(bi, offset) band_lower_left[(bi)].kmer_idx + (offset)

#define move_down(curr_band)                                                   \
    { curr_band.event_idx + 1, curr_band.kmer_idx }
#define move_right(curr_band)                                                  \
    { curr_band.event_idx, curr_band.kmer_idx + 1 }

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

#define BAND_ARRAY(r, c) ( bands[((r)*(ALN_BANDWIDTH)+(c))] )
#define TRACE_ARRAY(r, c) ( trace[((r)*(ALN_BANDWIDTH)+(c))] )

#define FROM_D  0
#define FROM_U  1
#define FROM_L  2


#define max_gap_threshold  50
#define bandwidth  ALN_BANDWIDTH
#define half_bandwidth  ALN_BANDWIDTH/2

#ifndef ALIGN_KERNEL_FLOAT
    #define min_average_log_emission  -5.0
    #define epsilon 1e-10
#else
    #define min_average_log_emission -5.0f
    #define epsilon 1e-10f
#endif



/************************kernels with 2D thread models*****************/



__global__ void align_kernel_pre_2d(char* read,
    int32_t* read_len, ptr_t* read_ptr,
    int32_t* n_events,
    ptr_t* event_ptr, model_t* models, uint32_t kmer_size,
    int32_t n_bam_rec,model_t* model_kmer_caches,float *bands1,uint8_t *trace1, EventKmerPair* band_lower_left1) {


    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int tid=blockIdx.x*blockDim.x+threadIdx.x;


    if (i < n_bam_rec) {
        char* sequence = &read[read_ptr[i]];
        int32_t sequence_len = read_len[i];
        //int32_t n_event = n_events[i];
        model_t* model_kmer_cache = &model_kmer_caches[read_ptr[i]];
        float *bands = &bands1[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace = &trace1[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left = &band_lower_left1[read_ptr[i]+event_ptr[i]];

        //int32_t n_events = n_event;
        int32_t n_kmers = sequence_len - kmer_size + 1;
        //fprintf(stderr,"n_kmers : %d\n",n_kmers);

        // transition penalties
        // float events_per_kmer = (float)n_events / n_kmers;
        // float p_stay = 1 - (1 / (events_per_kmer + 1));

        // setting a tiny skip penalty helps keep the true alignment within the adaptive band
        // this was empirically determined
        //double epsilon = 1e-10;
        // double lp_skip = log(epsilon);
        // double lp_stay = log(p_stay);
        // double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
        #ifndef ALIGN_KERNEL_FLOAT
            double lp_trim = log(0.01);
        #else
            float lp_trim = logf(0.01f);
        #endif

        // dp matrix
        //int32_t n_rows = n_events + 1;
        //int32_t n_cols = n_kmers + 1;
        //int32_t n_bands = n_rows + n_cols;

        // Initialize
        // Precompute k-mer ranks to avoid doing this in the inner loop

    // #ifdef  PRE_3D
    //     if(band_i<n_kmers && band_j==0){
    // #else
    //     if(band_i<n_kmers){
    // #endif

        if(tid==0){ //todo : can be optimised
            for (int32_t i = 0; i < n_kmers; ++i) {
                char* substring = &sequence[i];
                uint32_t kmer_ranks = get_kmer_rank(substring, kmer_size);
                model_kmer_cache[i] = models[kmer_ranks];
            }
        }

        if(tid<bandwidth){
            for (int32_t i = 0; i < 3; i++) {
                    BAND_ARRAY(i,tid) = -INFINITY;
                    //TRACE_ARRAY(i,tid) = 0;
            }
        }

        if(tid==0){
            // initialize range of first two bands
            band_lower_left[0].event_idx = half_bandwidth - 1;
            band_lower_left[0].kmer_idx = -1 - half_bandwidth;
            band_lower_left[1] = move_down(band_lower_left[0]);

            int start_cell_offset = band_kmer_to_offset(0, -1);
            assert(is_offset_valid(start_cell_offset));
            assert(band_event_to_offset(0, -1) == start_cell_offset);
            BAND_ARRAY(0,start_cell_offset) = 0.0f;

            // band 1: first event is trimmed
            int first_trim_offset = band_event_to_offset(1, 0);
            assert(kmer_at_offset(1, first_trim_offset) == -1);
            assert(is_offset_valid(first_trim_offset));
            BAND_ARRAY(1,first_trim_offset) = lp_trim;
            TRACE_ARRAY(1,first_trim_offset) = FROM_U;

            //int fills = 0;
        #ifdef DEBUG_ADAPTIVE
            fprintf(stderr, "[trim] bi: %d o: %d e: %d k: %d s: %.2lf\n", 1,
                    first_trim_offset, 0, -1, BAND_ARRAY(1,first_trim_offset);
        #endif

        }
    }
}


#define PROFILE 1

#define band_event_to_offset_shm(bi, ei) band_lower_left_shm[bi].event_idx - (ei)
#define band_kmer_to_offset_shm(bi, ki) (ki) - band_lower_left_shm[bi].kmer_idx

#define event_at_offset_shm(bi, offset) band_lower_left_shm[(bi)].event_idx - (offset)
#define kmer_at_offset_shm(bi, offset) band_lower_left_shm[(bi)].kmer_idx + (offset)

#define BAND_ARRAY_SHM(r, c) ( bands_shm[(r)][(c)] )
__global__ void
//__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
align_kernel_core_2d_shm(int32_t* read_len, ptr_t* read_ptr,
    event_t* event_table, int32_t* n_events1, ptr_t* event_ptr,
    scalings_t* scalings, int32_t n_bam_rec,model_t* model_kmer_caches, uint32_t kmer_size,
    float *band,uint8_t *traces, EventKmerPair* band_lower_lefts) {

    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int offset=blockIdx.x*blockDim.x+threadIdx.x;

    __shared__ float  bands_shm[3][ALN_BANDWIDTH];
    __shared__ EventKmerPair  band_lower_left_shm[3];

    if (i < n_bam_rec && offset<ALN_BANDWIDTH) {

        int32_t sequence_len = read_len[i];
        event_t* events = &event_table[event_ptr[i]];
        int32_t n_event = n_events1[i];
        scalings_t scaling = scalings[i];
        model_t* model_kmer_cache = &model_kmer_caches[read_ptr[i]];
        float *bands = &band[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace = &traces[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left = &band_lower_lefts[read_ptr[i]+event_ptr[i]];;

        // size_t n_events = events[strand_idx].n;
        int32_t n_events = n_event;
        int32_t n_kmers = sequence_len - kmer_size + 1;
        //fprintf(stderr,"n_kmers : %d\n",n_kmers);

        // transition penalties
        float events_per_kmer = (float)n_events / n_kmers;
        float p_stay = 1 - (1 / (events_per_kmer + 1));

        // setting a tiny skip penalty helps keep the true alignment within the adaptive band
        // this was empirically determined
        //double epsilon = 1e-10;

#ifndef ALIGN_KERNEL_FLOAT
        double lp_skip = log(epsilon);
        double lp_stay = log(p_stay);
        double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
        double lp_trim = log(0.01);
#else
        float lp_skip = logf(epsilon);
        float lp_stay = logf(p_stay);
        float lp_step = logf(1.0f - expf(lp_skip) - expf(lp_stay));
        float lp_trim = logf(0.01f);
#endif
        // dp matrix
        int32_t n_rows = n_events + 1;
        int32_t n_cols = n_kmers + 1;
        int32_t n_bands = n_rows + n_cols;


        BAND_ARRAY_SHM(0,offset) = BAND_ARRAY(2,offset);
        BAND_ARRAY_SHM(1,offset) = BAND_ARRAY(1,offset);
        BAND_ARRAY_SHM(2,offset) = BAND_ARRAY(0,offset);

        band_lower_left_shm[0] = band_lower_left[2];
        band_lower_left_shm[1] = band_lower_left[1];
        band_lower_left_shm[2] = band_lower_left[0];

        __syncthreads();

        // fill in remaining bands
        for (int32_t band_idx = 2; band_idx < n_bands; ++band_idx) {

            if(offset==0){
                // Determine placement of this band according to Suzuki's adaptive algorithm
                // When both ll and ur are out-of-band (ob) we alternate movements
                // otherwise we decide based on scores
                //float ll = BAND_ARRAY((band_idx - 1), 0);
                float ll = BAND_ARRAY_SHM((1), 0);
                //float ur = BAND_ARRAY((band_idx - 1),(bandwidth - 1));
                float ur = BAND_ARRAY_SHM((1),(bandwidth - 1));
                bool ll_ob = ll == -INFINITY;
                bool ur_ob = ur == -INFINITY;

                bool right = false;
                if (ll_ob && ur_ob) {
                    right = band_idx % 2 == 1;
                } else {
                    right = ll < ur; // Suzuki's rule
                }

                if (right) {
                    band_lower_left[band_idx] = band_lower_left_shm[0] =
                        move_right(band_lower_left_shm[1]);
                } else {
                    band_lower_left[band_idx] = band_lower_left_shm[0] =
                        move_down(band_lower_left_shm[1]);
                }
                // If the trim state is within the band, fill it in here
                int trim_offset = band_kmer_to_offset_shm(0, -1);
                if (is_offset_valid(trim_offset)) {
                    int32_t event_idx = event_at_offset_shm(0, trim_offset);
                    if (event_idx >= 0 && event_idx < n_events) {
                        //BAND_ARRAY(band_idx,trim_offset) = lp_trim * (event_idx + 1);
                        BAND_ARRAY_SHM(0,trim_offset) = lp_trim * (event_idx + 1);
                        TRACE_ARRAY(band_idx,trim_offset) = FROM_U;
                    } else {
                        //BAND_ARRAY(band_idx,trim_offset) = -INFINITY;
                        BAND_ARRAY_SHM(0,trim_offset) = -INFINITY;
                    }
                }
            }
            __syncthreads();

            // Get the offsets for the first and last event and kmer
            // We restrict the inner loop to only these values
            int kmer_min_offset = band_kmer_to_offset_shm(0, 0);
            int kmer_max_offset = band_kmer_to_offset_shm(0, n_kmers);
            int event_min_offset = band_event_to_offset_shm(0, n_events - 1);
            int event_max_offset = band_event_to_offset_shm(0, -1);

            int min_offset = MAX(kmer_min_offset, event_min_offset);
            min_offset = MAX(min_offset, 0);

            int max_offset = MIN(kmer_max_offset, event_max_offset);
            max_offset = MIN(max_offset, bandwidth);

            __syncthreads();

            if(offset>=min_offset && offset< max_offset) {

                int event_idx = event_at_offset_shm(0, offset);
                int kmer_idx = kmer_at_offset_shm(0, offset);

                //int32_t kmer_rank = kmer_ranks[kmer_idx];

                int offset_up = band_event_to_offset_shm(1, event_idx - 1);
                int offset_left = band_kmer_to_offset_shm(1, kmer_idx - 1);
                int offset_diag = band_kmer_to_offset_shm(2, kmer_idx - 1);

    #ifdef DEBUG_ADAPTIVE
                // verify loop conditions
                assert(kmer_idx >= 0 && kmer_idx < n_kmers);
                assert(event_idx >= 0 && event_idx < n_events);
                assert(offset_diag ==
                       band_event_to_offset_shm(2, event_idx - 1));
                assert(offset_up - offset_left == 1);
                assert(offset >= 0 && offset < bandwidth);
    #endif //DEBUG_ADAPTIVE

                float up = is_offset_valid(offset_up)
                               ? BAND_ARRAY_SHM(1,offset_up)
                               : -INFINITY;
                float left = is_offset_valid(offset_left)
                                 ? BAND_ARRAY_SHM(1,offset_left)
                                 : -INFINITY;
                float diag = is_offset_valid(offset_diag)
                                 ? BAND_ARRAY_SHM(2,offset_diag)
                                 : -INFINITY;

            #ifndef PROFILE
                float lp_emission = log_probability_match_r9(
                    scaling, model_kmer_cache, events, event_idx,kmer_idx);
                //fprintf(stderr, "lp emiision : %f , event idx %d, kmer rank %d\n", lp_emission,event_idx,kmer_rank);
            #else
                float unscaledLevel = events[event_idx].mean;
                float scaledLevel = unscaledLevel;
                model_t model = model_kmer_cache[kmer_idx];
                float gp_mean =
                    scaling.scale * model.level_mean + scaling.shift;
                float gp_stdv = model.level_stdv ; //scaling.var = 1;

                #ifdef  CACHED_LOG
                    float gp_log_stdv = model.level_log_stdv;
                #else
                    #ifndef ALIGN_KERNEL_FLOAT
                        float gp_log_stdv = log(gp_stdv); // scaling.log_var = log(1)=0;
                    #else
                        float gp_log_stdv = logf(gp_stdv); // scaling.log_var = log(1)=0;
                    #endif
                #endif

                float a = (scaledLevel - gp_mean) / gp_stdv;
                float lp_emission  = log_inv_sqrt_2pi - gp_log_stdv + (-0.5f * a * a);

            #endif


                float score_d = diag + lp_step + lp_emission;
                float score_u = up + lp_stay + lp_emission;
                float score_l = left + lp_skip;

                float max_score = score_d;
                uint8_t from = FROM_D;

                max_score = score_u > max_score ? score_u : max_score;
                from = max_score == score_u ? FROM_U : from;
                max_score = score_l > max_score ? score_l : max_score;
                from = max_score == score_l ? FROM_L : from;

    #ifdef DEBUG_ADAPTIVE
                fprintf(stderr,
                        "[adafill] offset-up: %d offset-diag: %d offset-left: %d\n",
                        offset_up, offset_diag, offset_left);
                fprintf(stderr, "[adafill] up: %.2lf diag: %.2lf left: %.2lf\n", up,
                        diag, left);
                fprintf(stderr,
                        "[adafill] bi: %d o: %d e: %d k: %d s: %.2lf f: %d emit: "
                        "%.2lf\n",
                        band_idx, offset, event_idx, kmer_idx, max_score, from,
                        lp_emission);
    #endif //DEBUG_ADAPTIVE
                //BAND_ARRAY(band_idx,offset) = max_score;
                BAND_ARRAY_SHM(0,offset) = max_score;
                TRACE_ARRAY(band_idx,offset) = from;
                //fills += 1;
            }



            __syncthreads();
            BAND_ARRAY(band_idx,offset) = BAND_ARRAY_SHM(0,offset);

            BAND_ARRAY_SHM(2,offset) = BAND_ARRAY_SHM(1,offset);
            BAND_ARRAY_SHM(1,offset) = BAND_ARRAY_SHM(0,offset);
            BAND_ARRAY_SHM(0,offset) = -INFINITY;

            if(offset==0){
                band_lower_left_shm[2]=band_lower_left_shm[1];
                band_lower_left_shm[1]=band_lower_left_shm[0];
            }


            __syncthreads();

        }
    }
}




//align post kernel
__global__ void align_kernel_post(AlignedPair* event_align_pairs,
    int32_t* n_event_align_pairs,
    int32_t* read_len, ptr_t* read_ptr,
    event_t* event_table, int32_t* n_events, ptr_t* event_ptr,
    scalings_t* scalings, int32_t n_bam_rec,model_t* model_kmer_caches, uint32_t kmer_size,
    float *bands1,uint8_t *trace1, EventKmerPair* band_lower_left1) {

    #ifndef WARP_HACK
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < n_bam_rec) {
    #else
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int i = tid/32;
        if (i < n_bam_rec && tid%32==0) {
    #endif
        AlignedPair* out_2 = &event_align_pairs[event_ptr[i] * 2];
        int32_t sequence_len = read_len[i];
        event_t* events = &event_table[event_ptr[i]];
        int32_t n_event = n_events[i];
        scalings_t scaling = scalings[i];
        model_t* model_kmer_cache = &model_kmer_caches[read_ptr[i]];
        float *bands = &bands1[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace = &trace1[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left = &band_lower_left1[read_ptr[i]+event_ptr[i]];;

        //fprintf(stderr, "%s\n", sequence);
        //fprintf(stderr, "Scaling %f %f", scaling.scale, scaling.shift);

        //size_t strand_idx = 0;
        //size_t k = 6;

        // size_t n_events = events[strand_idx].n;
        int32_t n_events = n_event;
        int32_t n_kmers = sequence_len - kmer_size + 1;
        //fprintf(stderr,"n_kmers : %d\n",n_kmers);
        // backtrack markers
        //const uint8_t FROM_D = 0;
        //const uint8_t FROM_U = 1;
        //const uint8_t FROM_L = 2;

        // qc
        //double min_average_log_emission = -5.0;
        //int max_gap_threshold = 50;

        // banding
        //int bandwidth = ALN_BANDWIDTH;
        //half_bandwidth = bandwidth / 2;

        // transition penalties
        float events_per_kmer = (float)n_events / n_kmers;
        float p_stay = 1 - (1 / (events_per_kmer + 1));

        // setting a tiny skip penalty helps keep the true alignment within the adaptive band
        // this was empirically determined
        //double epsilon = 1e-10;
#ifndef ALIGN_KERNEL_FLOAT
        double lp_skip = log(epsilon);
        double lp_stay = log(p_stay);
        double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
        double lp_trim = log(0.01);
#else
        float lp_skip = logf(epsilon);
        float lp_stay = logf(p_stay);
        float lp_step = logf(1.0f - expf(lp_skip) - expf(lp_stay));
        float lp_trim = logf(0.01f);
#endif
        // dp matrix
        int32_t n_rows = n_events + 1;
        int32_t n_cols = n_kmers + 1;
        int32_t n_bands = n_rows + n_cols;
        //
        // Backtrack to compute alignment
        //
        double sum_emission = 0;
        double n_aligned_events = 0;

        //>>>>>>>>>>>>>> New replacement begin
        // std::vector<AlignedPair> out;

        int outIndex = 0;
        //<<<<<<<<<<<<<<<<New Replacement over

        float max_score = -INFINITY;
        int curr_event_idx = 0;
        int curr_kmer_idx = n_kmers - 1;

        // Find best score between an event and the last k-mer. after trimming the remaining evnets
        for (int32_t event_idx = 0; event_idx < n_events; ++event_idx) {
            int band_idx = event_kmer_to_band(event_idx, curr_kmer_idx);

            //>>>>>>>New  replacement begin
            /*assert(band_idx < bands.size());*/

            assert(band_idx < n_bands);

            //<<<<<<<<New Replacement over
            int offset = band_event_to_offset(band_idx, event_idx);
            if (is_offset_valid(offset)) {
                float s =
                    BAND_ARRAY(band_idx,offset) + (n_events - event_idx) * lp_trim;
                if (s > max_score) {
                    max_score = s;
                    curr_event_idx = event_idx;
                }
            }
        }

    #ifdef DEBUG_ADAPTIVE
        fprintf(stderr, "[adaback] ei: %d ki: %d s: %.2f\n", curr_event_idx,
                curr_kmer_idx, max_score);
    #endif

        int curr_gap = 0;
        int max_gap = 0;
        while (curr_kmer_idx >= 0 && curr_event_idx >= 0) {
            // emit alignment
            //>>>>>>>New Repalcement begin
            assert(outIndex < n_events * 2);
            out_2[outIndex].ref_pos = curr_kmer_idx;
            out_2[outIndex].read_pos = curr_event_idx;
            outIndex++;
            // out.push_back({curr_kmer_idx, curr_event_idx});
            //<<<<<<<<<New Replacement over

    #ifdef DEBUG_ADAPTIVE
            fprintf(stderr, "[adaback] ei: %d ki: %d\n", curr_event_idx,
                    curr_kmer_idx);
    #endif
            // qc stats
            //>>>>>>>>>>>>>>New Replacement begin
            // char* substring = &sequence[curr_kmer_idx];
            // int32_t kmer_rank = get_kmer_rank(substring, kmer_size);
            // //<<<<<<<<<<<<<New Replacement over
            // float tempLogProb = log_probability_match_r9(
            //     scaling, models, events, curr_event_idx, kmer_rank);

            #ifndef PROFILE
                float tempLogProb = log_probability_match_r9(
                    scaling, model_kmer_cache, events, curr_event_idx,curr_kmer_idx);
                //fprintf(stderr, "lp emiision : %f , event idx %d, kmer rank %d\n", lp_emission,event_idx,kmer_rank);
            #else
                float unscaledLevel = events[curr_event_idx].mean;
                float scaledLevel = unscaledLevel;
                model_t model = model_kmer_cache[curr_kmer_idx];
                float gp_mean =
                    scaling.scale * model.level_mean + scaling.shift;
                float gp_stdv = model.level_stdv ; //scaling.var = 1;

                #ifdef  CACHED_LOG
                    float gp_log_stdv = model.level_log_stdv;
                #else
                    #ifndef ALIGN_KERNEL_FLOAT
                        float gp_log_stdv = log(gp_stdv); // scaling.log_var = log(1)=0;
                    #else
                        float gp_log_stdv = logf(gp_stdv); // scaling.log_var = log(1)=0;
                    #endif
                #endif

                float a = (scaledLevel - gp_mean) / gp_stdv;
                float tempLogProb  = log_inv_sqrt_2pi - gp_log_stdv + (-0.5f * a * a);

            #endif


            sum_emission += tempLogProb;
            //fprintf(stderr, "lp_emission %f \n", tempLogProb);
            //fprintf(stderr,"lp_emission %f, sum_emission %f, n_aligned_events %d\n",tempLogProb,sum_emission,outIndex);

            n_aligned_events += 1;

            int band_idx = event_kmer_to_band(curr_event_idx, curr_kmer_idx);
            int offset = band_event_to_offset(band_idx, curr_event_idx);
            assert(band_kmer_to_offset(band_idx, curr_kmer_idx) == offset);

            uint8_t from = TRACE_ARRAY(band_idx,offset);
            if (from == FROM_D) {
                curr_kmer_idx -= 1;
                curr_event_idx -= 1;
                curr_gap = 0;
            } else if (from == FROM_U) {
                curr_event_idx -= 1;
                curr_gap = 0;
            } else {
                curr_kmer_idx -= 1;
                curr_gap += 1;
                max_gap = MAX(curr_gap, max_gap);
            }
        }


#ifndef REVERSAL_ON_CPU
        //>>>>>>>>New replacement begin
        // std::reverse(out.begin(), out.end());
        int c;
        int end = outIndex - 1;
        for (c = 0; c < outIndex / 2; c++) {
            int ref_pos_temp = out_2[c].ref_pos;
            int read_pos_temp = out_2[c].read_pos;
            out_2[c].ref_pos = out_2[end].ref_pos;
            out_2[c].read_pos = out_2[end].read_pos;
            out_2[end].ref_pos = ref_pos_temp;
            out_2[end].read_pos = read_pos_temp;
            end--;
        }

        // if(outIndex>1){
        //   AlignedPair temp={out_2[0].ref_pos,out[0].read_pos};
        //   int i;
        //   for(i=0;i<outIndex-1;i++){
        //     out_2[i]={out_2[outIndex-1-i].ref_pos,out[outIndex-1-i].read_pos};
        //   }
        //   out[outIndex-1]={temp.ref_pos,temp.read_pos};
        // }
        //<<<<<<<<<New replacement over

        //>>>>>>>>>>>>>New replacement begin
        bool spanned = out_2[0].ref_pos == 0 &&
                    out_2[outIndex - 1].ref_pos == int(n_kmers - 1);

        //assert(spanned==spanned_before_rev);
        // bool spanned = out.front().ref_pos == 0 && out.back().ref_pos == n_kmers - 1;
        //<<<<<<<<<<<<<New replacement over
#else
        bool spanned = out_2[outIndex - 1].ref_pos == 0 &&
                    out_2[0].ref_pos == int(n_kmers - 1);
#endif
        // QC results
        double avg_log_emission = sum_emission / n_aligned_events;
        //fprintf(stderr,"sum_emission %f, n_aligned_events %f, avg_log_emission %f\n",sum_emission,n_aligned_events,avg_log_emission);

        //bool failed = false;
        if (avg_log_emission < min_average_log_emission || !spanned ||
            max_gap > max_gap_threshold) {
            //failed = true;
            //>>>>>>>>>>>>>New replacement begin
            outIndex = 0;
            // out.clear();
            //free(out_2);
            //out_2 = NULL;
            //<<<<<<<<<<<<<New replacement over
        }

        // free(kmer_ranks);
        // for (size_t i = 0; i < n_bands; i++) {
        //     free(bands[i]);
        //     free(trace[i]);
        // }
        // free(bands);
        // free(trace);
        // free(band_lower_left);
        //fprintf(stderr, "ada\t%s\t%s\t%.2lf\t%zu\t%.2lf\t%d\t%d\t%d\n", read.read_name.substr(0, 6).c_str(), failed ? "FAILED" : "OK", events_per_kmer, sequence.size(), avg_log_emission, curr_event_idx, max_gap, fills);
        //outSize=outIndex;
        //if(outIndex>500000)fprintf(stderr, "Max outSize %d\n", outIndex);
        n_event_align_pairs[i] = outIndex;

    }
}
